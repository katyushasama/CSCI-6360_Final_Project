#include "hip/hip_runtime.h"
#include<cstdio>
#include "matrix.h"

//Matrix Vector Multiplication
#define NTHREADS_X 32
#define NTHREADS_Y 32

__global__ void MatrixMultiplication(float *a, float *b, float *c, int a_ncolumns, int c_nlines, int c_ncolumns,int rank)
{  	
    int column = blockIdx.x * blockDim.x + threadIdx.x;
    int line =  blockIdx.y * blockDim.y + threadIdx.y;

    if (column  >= c_ncolumns || line >= c_nlines)
        return;

    int i = 0;
    float sum = 0.0;

    int beginA = a_ncolumns * line;
    int beginB = column;

    for (i = 0; i < a_ncolumns; i++)
    {
        sum += a[beginA + i] * b[i * c_ncolumns + beginB];
    }

    c[line * c_ncolumns + column] = sum;
}

extern "C" 
Matrix cudaInit(int myrank, const float* MyMatrixA, const float* MatrixB, int ColsNo, const int elements, const int size, const size_t matrixbsize){
    hipError_t cE;
    int cudaDeviceCount;
    cE = hipGetDeviceCount(&cudaDeviceCount);
    if(cE != hipSuccess){
        printf("Unable to determine cuda device count, error is %d, count is %d\n", cE, cudaDeviceCount);
        exit(-1);
    }
    cE = hipSetDevice(myrank % cudaDeviceCount);
    if(cE != hipSuccess){
        printf("Unable to have rank %d set to cuda device %d, error is %d \n", myrank, (myrank % cudaDeviceCount), cE);
        exit(-1);
    }
    Matrix M;
    M.sizeA = size;
    M.sizeB = matrixbsize;

	//Allocating the Memory on the device memory
	hipMallocManaged( (void **)&M.MatrixA, size * ColsNo * sizeof(float) );
	hipMallocManaged( (void **)&M.MatrixB, matrixbsize*sizeof(float) );
	hipMallocManaged( (void **)&M.result, elements * sizeof(float) );

	//Copying the data from host to device
	hipMemcpy( (void *)M.MatrixA, (void *)MyMatrixA, size * ColsNo * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( (void *)M.MatrixB, (void *)MatrixB,  matrixbsize*sizeof(float), hipMemcpyHostToDevice );
    return M;
}

extern "C" 
void cudaReduce(float* MyResultMatrix, Matrix* M, int elements, int RowsNo, int ColsNo, int RowsNo2, int ColsNo2, int size, int rank){
        dim3 blocks = dim3(
                    (int) std::ceil( (double) ColsNo/ NTHREADS_X ),
                    (int) std::ceil ( (double) (RowsNo/size)/ NTHREADS_Y ),
                    1
                );

    dim3 threads = dim3(
                        NTHREADS_X,
                        NTHREADS_Y,
                        1
                    );
    MatrixMultiplication<<<blocks, threads>>>(M->MatrixA, M->MatrixB, M->result, RowsNo2,RowsNo/size,ColsNo,rank);	
	hipMemcpy( (void *)MyResultMatrix, (void *)M->result, elements * sizeof(float), hipMemcpyDeviceToHost );
	hipDeviceSynchronize();
}